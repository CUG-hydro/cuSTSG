//******************************************************************
//cuSTSG is used to reconstruct high-quality NDVI time series data(MODIS/SPOT) based on STSG
//
//This procedure cuSTSG is the source code for the first version of cuSTSG.
//This is a parallel computing code using GPU.
//
//Coded by Yang Xue
//******************************************************************

#include "Filter.h"

#include "gdal_priv.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include <iostream>
#include <algorithm>
#include <fstream>

using namespace std;

int main(int argc, char *argv[])
{
	GDALAllRegister();
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	GDALDriver *pDriver = GetGDALDriverManager()->GetDriverByName("GTIFF");
	char **ppszOptions = NULL;
	ppszOptions = CSLSetNameValue(ppszOptions, "BIGTIFF", "IF_NEEDED");

	//parameters
	if (argc != 2)
	{
		cout << "No parameter file!" << endl;
		return 1;
	}

	ifstream parameter(argv[1]);
	if (!parameter)
	{
		cout << "Can't open parameter file!" << endl;
		return 1;
	}

	int* Years = nullptr;
	string NDVI_path, Reliability_path, STSG_Test_path;
	float cosyear, sampcorr;
	int win_year, win, snow_address, n_Years;
	string par;
	while (getline(parameter, par))
	{
		if (par.substr(0, 2) == "//" || par == "")
			continue;

		for (int i = 0; i < par.size(); )
		{
			if (isspace(par[i]))
				par.erase(i,1);
			else
				i++;
		}
		if (par.substr(0, par.find("=")) == "Years")
		{
			vector<int> year;
			while (par.rfind(",") < par.size())
			{
				year.push_back(stoi(par.substr(par.rfind(",") + 1)));
				par = par.substr(0, par.rfind(","));
			}
			year.push_back(stoi(par.substr(par.rfind("=") + 1)));

			n_Years = year.size();
			Years = new int[n_Years];
			for (int i = 0; i < n_Years; i++)
				Years[i] = year[n_Years - i - 1];
		}
		else if (par.substr(0, par.find("=")) == "NDVI_path")
			NDVI_path = par.substr(par.find("=") + 1);
		else if (par.substr(0, par.find("=")) == "Reliability_path")
			Reliability_path = par.substr(par.find("=") + 1);
		else if (par.substr(0, par.find("=")) == "STSG_Test_path")
			STSG_Test_path = par.substr(par.find("=") + 1);
		else if (par.substr(0, par.find("=")) == "cosyear")
			cosyear = stof(par.substr(par.find("=") + 1));
		else if (par.substr(0, par.find("=")) == "win_year")
			win_year = stoi(par.substr(par.find("=") + 1));
		else if (par.substr(0, par.find("=")) == "win")
			win = stoi(par.substr(par.find("=") + 1));
		else if (par.substr(0, par.find("=")) == "sampcorr")
			sampcorr = stof(par.substr(par.find("=") + 1));
		else if (par.substr(0, par.find("=")) == "snow_address")
			snow_address = stoi(par.substr(par.find("=") + 1));
	}
	parameter.close();

	int n_Device;
	hipGetDeviceCount(&n_Device);
	hipDeviceProp_t prop;
	for (int i = 0; i < n_Device; i++)
	{
		hipGetDeviceProperties(&prop, i);
		cout << "Device " << i << "'s Property:" << endl;
		cout << "Name:" << prop.name << endl;
		cout << "TotalGlobalMem:" << prop.totalGlobalMem / 1024 / 1024 << " MB" << endl;
		cout << "SharedMemPerBlock:" << prop.sharedMemPerBlock / 1024 << " KB" << endl;
		cout << "MaxThreadsPerBlock:" << prop.maxThreadsPerBlock << endl;
		cout << "MaxGridSize:" << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;
		cout << "WarpSize:" << prop.warpSize << endl;
	}

	//cuSTSG
	vector<GDALDataset*> NDVI(n_Years);
	vector<GDALDataset*> QA(n_Years);
	int n_X, n_Y, n_B;
	GDALDataType type_NDVI, type_QA;
	for (int i = 0; i < n_Years; i++)
	{
		string FileName = NDVI_path + to_string(Years[i]);
		NDVI[i] = (GDALDataset*)GDALOpen(FileName.c_str(), GA_ReadOnly);
		if (i == 0)
		{
			n_X = NDVI[i]->GetRasterXSize();
			n_Y = NDVI[i]->GetRasterYSize();
			n_B = NDVI[i]->GetRasterCount();
			type_NDVI = NDVI[i]->GetRasterBand(1)->GetRasterDataType();
		}

		FileName = Reliability_path + to_string(Years[i]);
		QA[i] = (GDALDataset*)GDALOpen(FileName.c_str(), GA_ReadOnly);
		if (i == 0)
			type_QA = QA[i]->GetRasterBand(1)->GetRasterDataType();
	}

	cout << "Start: cuSTSG" << endl;
	size_t PerYSize = n_X*n_B *(n_Years * sizeof(short) + n_Years * sizeof(unsigned char) + 2 * n_Years * sizeof(float) + sizeof(int) + sizeof(float) + n_Years * sizeof(float)) + n_X*(2 * win + 1)*(2 * win + 1) *(7 * sizeof(float) + 3 * sizeof(int));
	if (prop.totalGlobalMem <= 2 * win*n_X*n_B* (n_Years * sizeof(short) + n_Years * sizeof(unsigned char) + 2 * n_Years * sizeof(float) + sizeof(float)) + n_X*n_Y*n_B*n_Years*sizeof(float))
	{
		cout << "Size of vector_out is larger than totalGlobalMem!" << endl;
		return 1;
	}

	size_t PerStep = (prop.totalGlobalMem - 2 * win*n_X*n_B* (n_Years * sizeof(short) + n_Years * sizeof(unsigned char) + 2 * n_Years * sizeof(float) + sizeof(float)) - n_X*n_Y*n_B*n_Years*sizeof(float)) / PerYSize;
	int Loops = 1;
	if (PerStep < n_Y)
	{
		Loops = n_Y / PerStep + 1;
		PerStep = n_Y / Loops + 1;
	}

	float *d_vector_out;
	size_t nBytes = n_X*n_Y*n_B*n_Years * sizeof(float);
	hipMalloc((void**)&d_vector_out, nBytes);
	hipMemset((void*)d_vector_out, 0, nBytes);
	nBytes = win*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
	float *res = (float*)malloc(nBytes);  //(res_cosyear; Slope_res; Intercept_res; new_corr_similar_res;)
	memset((void*)res, 0, nBytes);
	int last_Buffer_Dn = 0;
	for (int i = 1, StartY = 0; i <= Loops&&StartY < n_Y; i++, StartY += PerStep)
	{
		cout << "Loops " << i << endl;
		if (i == Loops)
			PerStep = n_Y - StartY;

		int Buffer_Up = 0;
		int Buffer_Dn = 0;
		if (StartY + PerStep < n_Y - win)
			Buffer_Dn = win;
		else
			Buffer_Dn = n_Y - PerStep - StartY;
		if (StartY >= win)
			Buffer_Up = win;
		else
			Buffer_Up = StartY;

		int blkwidth = 16;
		int blkheight = 16;
		dim3 blocks(blkwidth, blkheight);
		dim3 grids(n_X % blkwidth == 0 ? n_X / blkwidth : n_X / blkwidth + 1, (PerStep + Buffer_Up + Buffer_Dn) % blkheight == 0 ? (PerStep + Buffer_Up + Buffer_Dn) / blkheight : (PerStep + Buffer_Up + Buffer_Dn) / blkheight + 1);

		short *img_NDVI = new short[(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years];
		unsigned char *img_QA = new unsigned char[(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years];
		for (int i = 0; i < n_Years; i++)
		{
			NDVI[i]->RasterIO(GF_Read, 0, StartY - Buffer_Up, n_X, (PerStep + Buffer_Up + Buffer_Dn), &img_NDVI[i*(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B], n_X, (PerStep + Buffer_Up + Buffer_Dn), type_NDVI, n_B, nullptr, 0, 0, 0);
			QA[i]->RasterIO(GF_Read, 0, StartY - Buffer_Up, n_X, (PerStep + Buffer_Up + Buffer_Dn), &img_QA[i*(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B], n_X, (PerStep + Buffer_Up + Buffer_Dn), type_QA, n_B, nullptr, 0, 0, 0);
		}

		short *d_imgNDVI;
		nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years * sizeof(short);
		hipMalloc((void**)&d_imgNDVI, nBytes);
		hipMemcpy((void*)d_imgNDVI, (void*)img_NDVI, nBytes, hipMemcpyHostToDevice);
		unsigned char *d_imgQA;
		nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years * sizeof(unsigned char);
		hipMalloc((void**)&d_imgQA, nBytes);
		hipMemcpy((void*)d_imgQA, (void*)img_QA, nBytes, hipMemcpyHostToDevice);
		float *d_img_NDVI, *d_img_QA;
		nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years *sizeof(float);
		hipMalloc((void**)&d_img_NDVI, nBytes);
		hipMalloc((void**)&d_img_QA, nBytes);
		hipMemset((void*)d_img_NDVI, 0, nBytes);
		hipMemset((void*)d_img_QA, 0, nBytes);
		float *d_NDVI_Reference, *d_res;
		nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B *  sizeof(float);
		hipMalloc((void**)&d_NDVI_Reference, nBytes);
		hipMemset((void*)d_NDVI_Reference, 0, nBytes);
		nBytes = (PerStep + Buffer_Dn)*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
		hipMalloc((void**)&d_res, nBytes);
		hipMemset((void*)d_res, 0, nBytes);
		nBytes = last_Buffer_Dn*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
		hipMemcpy((void*)d_res, (void*)res, nBytes, hipMemcpyHostToDevice);

		int *d_res_vec_res1;
		nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B * sizeof(int);
		hipMalloc((void**)&d_res_vec_res1, nBytes);
		hipMemset((void*)d_res_vec_res1, 0, nBytes);
		float *d_vector_in, *d_res_3;
		nBytes = PerStep*n_X* n_B * sizeof(float);
		hipMalloc((void**)&d_vector_in, nBytes);
		hipMemset((void*)d_vector_in, 0, nBytes);
		nBytes = PerStep*n_X*(2 * win + 1)*(2 * win + 1) * 3 * sizeof(float);
		hipMalloc((void**)&d_res_3, nBytes);//(slope_intercept(2);corr_similar;)
		hipMemset((void*)d_res_3, 0, nBytes);
		int *d_index;
		nBytes = PerStep*n_X*(2 * win + 1)*(2 * win + 1) * 3 * sizeof(int);
		hipMalloc((void**)&d_index, nBytes);//(similar_index(2);new_corr;)
		hipMemset((void*)d_index, 0, nBytes);
		hipDeviceSynchronize();

		Short_to_Float << <grids, blocks >> >(d_imgNDVI, d_imgQA, n_X, (PerStep + Buffer_Up + Buffer_Dn), n_B, n_Years, d_img_NDVI, d_img_QA);
		hipDeviceSynchronize();

		Generate_NDVI_reference << <grids, blocks >> >(cosyear, win_year, d_img_NDVI, d_img_QA, n_X, (PerStep + Buffer_Up + Buffer_Dn), n_B, n_Years, d_NDVI_Reference, d_res_3, d_res_vec_res1);
		hipDeviceSynchronize();

		nBytes = PerStep*n_X*(2 * win + 1)*(2 * win + 1) * 3 * sizeof(float);
		hipMemset((void*)d_res_3, 0, nBytes);
		hipDeviceSynchronize();

		Compute_d_res << <grids, blocks >> >(d_img_NDVI, d_img_QA, d_NDVI_Reference, StartY, n_Y, Buffer_Up, Buffer_Dn, n_X, (PerStep + Buffer_Up + Buffer_Dn), n_B, n_Years, win, d_res);
		hipDeviceSynchronize();

		STSG_filter << <grids, blocks >> >(d_img_NDVI, d_img_QA, d_NDVI_Reference, StartY, n_Y, Buffer_Up, Buffer_Dn, n_X, PerStep, n_B, n_Years, win, sampcorr, snow_address, d_vector_out, d_vector_in, d_res, d_res_3, d_index);
		hipDeviceSynchronize();

		nBytes = win*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
		memset((void*)res, 0, nBytes);
		nBytes = Buffer_Dn*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
		hipMemcpy((void*)res, (void*)&d_res[(PerStep + Buffer_Dn - win)*n_X*(2 * win + 1)*(2 * win + 1) * 4], nBytes, hipMemcpyDeviceToHost);
		last_Buffer_Dn = Buffer_Dn;
		hipDeviceSynchronize();

		delete[] img_NDVI;
		delete[] img_QA;
		hipFree((void*)d_imgNDVI);
		hipFree((void*)d_imgQA);
		hipFree((void*)d_img_NDVI);
		hipFree((void*)d_img_QA);
		hipFree((void*)d_NDVI_Reference);
		hipFree((void*)d_res);
		hipFree((void*)d_res_vec_res1);
		hipFree((void*)d_vector_in);
		hipFree((void*)d_res_3);
		hipFree((void*)d_index);
	}
	free((void*)res);
	hipDeviceSynchronize();

	float *vector_out = new float[n_X*n_Y*n_B*n_Years];
	nBytes = n_X*n_Y*n_B*n_Years* sizeof(float);
	hipMemcpy((void*)vector_out, (void*)d_vector_out, nBytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree((void*)d_vector_out);
	hipDeviceSynchronize();

	GDALDataset *File = pDriver->Create(STSG_Test_path.c_str(), n_X, n_Y, n_B*n_Years, GDT_Float32, ppszOptions);
	File->RasterIO(GF_Write, 0, 0, n_X, n_Y, vector_out, n_X, n_Y, GDT_Float32, n_B*n_Years, 0, 0, 0, 0);
	GDALClose(File);
	delete[] vector_out;
	
	return 0;
}